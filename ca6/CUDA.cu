
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
typedef unsigned char byte;

void toGreyScale(byte *input, byte *output, int h, int w, int ch) {
    int i, j;
    for (i = 0; i < h; i++) {
        for (j = 0; j < w; j++) {
            int ind = i * w * ch + j * ch;
            byte res = input[ind + 0] * 0.2989 + input[ind + 1] * 0.5870 + input[ind + 2] * 0.1140;
            output[i * w + j] = res;
        }
    }
}


void prewitt(byte *input, byte *output, int h, int w) {
    int x, y;
    byte *img = input;
    for (y = 1; y < h - 1; y++) {
        for (x = 1; x < w - 1; x++) {
            int vKer = 0, hKer = 0;

            vKer = img[(y-1)*w+(x-1)] * -1 + img[(y-1)*w+x] * -1 + img[(y-1)*w+(x+1)] * -1 +
                   img[(y+1)*w+(x-1)] *  1 + img[(y+1)*w+x] *  1 + img[(y+1)*w+(x+1)] *  1;

            hKer = img[(y-1)*w+(x-1)] * -1 + img[(y-1)*w+(x+1)] *  1 +
                   img[y*w+(x-1)] * -1 + img[y*w+(x+1)] *  1 +
                   img[(y+1)*w+(x-1)] * -1 + img[(y+1)*w+(x+1)] *  1;

            int gradient = (int)sqrt(hKer * hKer + vKer * vKer);
            gradient = gradient > 255 ? 255 : gradient;

            output[y * w + x] = (byte)gradient;
        }
    }
}
