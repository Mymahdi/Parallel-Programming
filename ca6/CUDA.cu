#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <opencv2/opencv.hpp>


__constant__ float sobelX[3][3] = {
    {-1, 0, 1},
    {-2, 0, 2},
    {-1, 0, 1}
};

__constant__ float sobelY[3][3] = {
    {-1, -2, -1},
    { 0,  0,  0},
    { 1,  2,  1}
};


int main() {
    cv::Mat image = cv::imread("images/Lenna.png", cv::IMREAD_GRAYSCALE);
    if (image.empty()) {
        std::cerr << "Error: Unable to load image!" << std::endl;
        return -1;
    }

    int width = image.cols;
    int height = image.rows;
    size_t imgSize = width * height * sizeof(unsigned char);

    unsigned char* h_input = image.data;
    unsigned char* h_output = (unsigned char*)malloc(imgSize);

    unsigned char *d_input, *d_output;
    hipMalloc(&d_input, imgSize);
    hipMalloc(&d_output, imgSize);
    hipMemcpy(d_input, h_input, imgSize, hipMemcpyHostToDevice);

    std::cout << "Image loaded successfully!" << std::endl;
    return 0;
}
