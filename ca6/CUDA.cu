
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 16

__global__ void applyKernel(const float* input, float* output, int padded_height, int padded_width, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int image_height = padded_height - 2;
    int image_width = padded_width - 2;

    if (x >= image_width || y >= image_height) return;

    float kernel[9] = {
         0,  1,  0,
         1, -2,  1,
         0,  1,  0
    };

    for (int c = 0; c < channels; ++c) {
        float value = 0.0f;

        value += input[((y + 0) * padded_width + (x + 0)) * channels + c] * kernel[0];
        value += input[((y + 0) * padded_width + (x + 1)) * channels + c] * kernel[1];
        value += input[((y + 0) * padded_width + (x + 2)) * channels + c] * kernel[2];
        value += input[((y + 1) * padded_width + (x + 0)) * channels + c] * kernel[3];
        value += input[((y + 1) * padded_width + (x + 1)) * channels + c] * kernel[4];
        value += input[((y + 1) * padded_width + (x + 2)) * channels + c] * kernel[5];
        value += input[((y + 2) * padded_width + (x + 0)) * channels + c] * kernel[6];
        value += input[((y + 2) * padded_width + (x + 1)) * channels + c] * kernel[7];
        value += input[((y + 2) * padded_width + (x + 2)) * channels + c] * kernel[8];

        output[(y * image_width + x) * channels + c] = value;
    }
}
